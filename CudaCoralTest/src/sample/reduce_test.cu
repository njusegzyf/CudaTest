#include "sample_test.h"

#include <stdio.h>

#include <functional>
#include <memory>
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "../util/cuda_smart_pointer.cuh"
#include "reduce.cuh"

//#define THREAD_SIZE (2048*2048)
static constexpr size_t THREAD_SIZE = 2048 * 2048;
//#define THREADS_PER_BLOCK 1024
static constexpr size_t THREADS_PER_BLOCK = 1024;
// Note: we will get an error if the BLOCK_SIZE is not a const
static constexpr size_t BLOCK_SIZE = (THREAD_SIZE + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK;

// @see http://en.cppreference.com/w/cpp/chrono/duration
using std::chrono::system_clock;
using std::cout;
using std::endl;

namespace cudatest {

  void runReduceTest() {

    hipError_t cudaStatus;
    int inputArrayByteSize = THREAD_SIZE * sizeof(float);
    int outputArrayByteSize = THREAD_SIZE / THREADS_PER_BLOCK * sizeof(float);

    /* allocate space for host memory */
    // float *h_xs, *h_outs;
    // h_xs = (float*)malloc(inputDoubleArrayByteSize);
    // h_outs = (float*)malloc(outputBoolArrayByteSize);
    auto h_xs = std::make_unique<float[]>(THREAD_SIZE);
    auto h_outs = std::make_unique<float[]>(BLOCK_SIZE);

    for (int i = 0; i < THREAD_SIZE; i++) {
      h_xs[i] = float(i);
    }

    /* allocate space for device copies */
    // float *d_xs, *d_outs;
    // cudaStatus = hipMalloc((void **)&d_xs, inputDoubleArrayByteSize);
    // cudaStatus = hipMalloc((void **)&d_outs, outputBoolArrayByteSize);
    // cudaStatus = hipMemset(d_outs, 0, outputBoolArrayByteSize);
    auto d_xs = cudautil::makeUniqueDeviceMemory<float>(THREAD_SIZE);
    auto d_outs = cudautil::makeUniqueDeviceMemory<float>(BLOCK_SIZE);
    cudaStatus = hipMemset(d_outs.get(), 0, outputArrayByteSize);

    /* copy inputs to device */
    hipMemcpy(d_xs.get(), h_xs.get(), inputArrayByteSize, hipMemcpyHostToDevice);

    // record kernel start time
    auto reduceGlobalStartTime = system_clock::now(); // of type system_clock::time_point

    /* launch the kernel on the GPU */
    cudatest::reduceGlobalKernel << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_xs.get(), d_outs.get());

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      // goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors after launching the kernel.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel, error : %s. \n",
              cudaStatus,
              hipGetErrorString(cudaStatus));
      // goto Error;
    }

    auto reduceGlobalEndTime = system_clock::now(); // of type system_clock::time_point

                                                    /* copy result back to host */
    hipMemcpy(h_outs.get(), d_outs.get(), outputArrayByteSize, hipMemcpyDeviceToHost);

    if (hipSuccess == cudaStatus) {
      system_clock::time_point cudaEndTime = system_clock::now();
      cout << "CUDA use: " << std::chrono::duration_cast<std::chrono::microseconds>(reduceGlobalEndTime - reduceGlobalStartTime).count() << " microseconds\n";
    }

    /* clean up */
    //free(h_xs);
    //free(h_outs);
    //hipFree(d_xs);
    //hipFree(d_outs);
  }
}
